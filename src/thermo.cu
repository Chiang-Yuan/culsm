#include "hip/hip_runtime.h"
#include "thermo.h"

// // host vectors

double* h_parsum_pe;

// // device vectors

double* d_parsum_pe; 		// partial sum of potential energy

__global__ void reduce_pe(
	double* x, float* k, float* r0, int* atom_i, int* atom_j,
	double* parsum_pe, // partial sum of potential energy
	int nbonds
)
{
	extern __shared__ double cache[];

	int i = blockIdx.x * blockDim.x + threadIdx.x;
	
	double temp = 0;

	while (i < nbonds) {

		int a3_i = atom_i[i] * 3;
        int a3_j = atom_j[i] * 3;
        
        double rij = sqrt(pow(x[a3_j] - x[a3_i], 2) + 
			pow(x[a3_j + 1] - x[a3_i + 1], 2) + 
			pow(x[a3_j + 2] - x[a3_i + 2], 2));

		temp += 1.0/2.0*k[i]*pow(rij - r0[i], 2);
		
		i += gridDim.x * blockDim.x;
	}
	cache[threadIdx.x] = temp;

	__syncthreads();

	// parallel reduction

	int ihalf = blockDim.x/2;
	while (ihalf != 0) {
		if (threadIdx.x < ihalf) cache[threadIdx.x] += cache[threadIdx.x + ihalf];
		__syncthreads();
		ihalf /= 2;
	}
	if (threadIdx.x == 0) parsum_pe[blockIdx.x] = cache[threadIdx.x];
	__syncthreads();
}

Thermo::Thermo(Error *error_)
{
	error = error_;
}

Thermo::~Thermo()
{
}

int Thermo::set(int nthermo_)
{
    nthermo = nthermo_;
	return 0;
}

double Thermo::pe(System & sys)
{
	// TODO: parallel reduction
	// double energy = 0;

	// TODO: other potentials (non-bonded, angles, etc)

	// bonds

	// bigint b;
	// int btype;
	// bigint ai;
	// bigint aj;

	// double k;
	// double r0;
	// double rij;	

	int threadsPerBlockforBonds = BLOCK_SIZE;
    int blocksPerGridforBonds = (sys.nbonds + threadsPerBlockforBonds - 1)/threadsPerBlockforBonds;

	dim3 dimBlockforBond(threadsPerBlockforBonds, 1, 1);
    dim3 dimGridforBond(blocksPerGridforBonds, 1, 1);

	int bond_double_sm_size = blocksPerGridforBonds*sizeof(double);
	h_parsum_pe = (double*)calloc(blocksPerGridforBonds, sizeof(double));

	hipMalloc((void**)&d_parsum_pe, bond_double_sm_size);

	hipMemcpy(d_parsum_pe, h_parsum_pe, bond_double_sm_size, hipMemcpyHostToDevice);

	int sm = threadsPerBlockforBonds*sizeof(double);
	reduce_pe<<<dimGridforBond, dimBlockforBond, sm>>>(
		d_x, d_k, d_r0, d_atom_i, d_atom_j,
		d_parsum_pe, // partial sum of potential energy
		sys.nbonds
    );
	hipMemcpy(h_parsum_pe, d_parsum_pe, bond_double_sm_size, hipMemcpyDeviceToHost);

	// Add the partial sum of all blocks
	double sum_pe = 0;
	for (int i = 0; i < blocksPerGridforBonds; i++) {
		sum_pe += h_parsum_pe[i];
	}

	// for (b = 0; b < sys.nbonds; b++) {
	// 	btype = sys.bond_type[b];
	// 	// k = sys.bondTypes[itype].coeff[0];
	// 	k = h_k[b];
	// 	r0 = sys.bondTypes[btype - 1].coeff[1];
		
	// 	ai = sys.atom_i[b];
	// 	aj = sys.atom_j[b];
		
	// 	rij = sqrt(pow(sys.x[ai*3] - sys.x[aj*3],2) + 
	// 		pow(sys.x[ai*3 + 1] - sys.x[aj*3 + 1],2) +
	// 		pow(sys.x[ai*3 + 2] - sys.x[aj*3 + 2],2));

	// 	energy += 1.0/2.0*k*pow(rij - r0, 2);
	// }

	return sum_pe;
}

double Thermo::ke(System & sys)
{
	// TODO: parallel reduction
	double energy = 0;

	bigint a;
	int type;
	double mass;
	double v2;

	for (a = 0; a < sys.natoms; a++) {
		type = sys.type[a];
		mass = sys.atomTypes[type - 1].mass;
		
		v2 = pow(sys.v[a*3], 2) + 
			pow(sys.v[a*3 + 1], 2) +
			pow(sys.v[a*3 + 2], 2);

		energy += 1.0/2.0*mass*v2;
	}

	return energy;
}

int Thermo::write_thermo(int timestep, System & sys)
{
	printf("%10d \t %16.9e \t %16.9e\n", 
		timestep, pe(sys), ke(sys));

	return 0;
}


